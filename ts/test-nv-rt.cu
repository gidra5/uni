
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello_kernel()
{
  printf("Hello World from GPU!\n");
}

extern "C" int cuda_hello()
{
  cuda_hello_kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}

// int main()
// {
//   return cuda_hello();
// }